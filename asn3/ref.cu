#include "hip/hip_runtime.h"
#include "ref.h"
// #include "hip/hip_runtime.h"
// #include "hip/hip_runtime_api.h"

#define TPB 16

 __global__
void scale_row(float *MAT, int pivot)
{
    int thread_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int thread_y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int tid = thread_y * N + thread_x;
    
    if (thread_y == pivot && thread_y < M && thread_x < N)
    {
        /* Assuming pivot col == pivot row always */
        int pivot_idx = pivot * N + pivot;
        MAT[tid] /= MAT[pivot_idx];
    }
}

__global__
void subtract_row(float *MAT, int pivot)
{
    int thread_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int thread_y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int tid = thread_y * N + thread_x;

    if (thread_y != pivot && thread_y < M && thread_x < N)
    {
        int pivot_idx = pivot * N + pivot;

        /* The factor to divide by */
        float f = MAT[thread_y * N] / MAT[pivot_idx];
        MAT[tid] -= f * MAT[pivot_idx];
    }
}

void ref_cuda(float *MAT)
{

    /* Allocate memory for the device */
    float *MATD;
    hipMalloc(&MATD, sizeof(float) * M * N);
    hipMemcpy(MATD, (void *) MAT, sizeof(float) * M * N, hipMemcpyHostToDevice);

    /* Loop through rows */
    for (int row = 0; row < M; row++)
    {
        /* Assuming square positive matrix always */
        int col = row;

        /* Block size is the number of blocks required 
         * 
         * So if we have 32 elements in the row and TPB is 16, we need
         * 2 blocks to process this row
         *
         * Optimization can be made since we don't need to process
         * all the elements before the pivot, since
         * we can assume they're zeros
         *
         * Normally we just do M / TPB
         * But this is modified so that we always have at least 1 block
         */
        int elements_to_process = N - col;
        int block_size = (elements_to_process - 1) / TPB + 1;
        scale_row<<<block_size, TPB>>>(MATD, row);
        hipDeviceSynchronize();

        /* Block size is now the remining elements */
        elements_to_process = (N - row) * (M - col);
        block_size = (elements_to_process - 1) / TPB + 1;
        subtract_row<<<block_size, TPB>>>(MATD, row);
        hipDeviceSynchronize();
    }

    /* Copy back from device to host */
    hipMemcpy(MAT, (void *) MATD, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    /* Free device memroy */
    hipFree(MATD);
}

int main(void)
{
    /* Malloc matrices */
    MAT   = (float *) malloc(sizeof(float) * N * M);
    MAT_B = (float *) malloc(sizeof(float) * N * M);

    printf("(M x N)=(%d x %d)\n", M, N);
    init_array(MAT);
    memcpy(MAT_B, MAT, sizeof(float) * N * M);
    print_mat(MAT);

    /* Run single threaded */
    printf("Running serial . . .\n");
    struct timeval begin, end;
    gettimeofday(&begin, 0);
    ref_old_noswap(MAT_B);
    gettimeofday(&end, 0);
    double time_serial = (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1e-6;
    print_mat(MAT_B);

    /* Run parallel ref */
    printf("Running parallel . . .\n");
    gettimeofday(&begin, 0);
    ref_cuda(MAT);
    gettimeofday(&end, 0);
    double time_parallel = (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1e-6;
    print_mat(MAT);

    /* Run verification (if enabled) */
    #ifdef RUN_VERIF
    printf("Running verification . . .\n");
    int errors = verify_ref(MAT, MAT_B);
    printf("MISMATCH=%d\n", errors);
    #endif

    printf("SERIAL TIME=%.6e s\n", time_serial);
    printf("PARALL TIME=%.6e s\n", time_parallel);

    /* Make sure we exit pthread */
    // pthread_exit(NULL);

    return 0;
}
