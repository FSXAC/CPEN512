#include "hip/hip_runtime.h"
#include "ref.h"

 __global__
void scale_row(float *MAT, int pivot)
{
    int tid = blockIdx.y * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        /* Assuming pivot col == pivot row always */
        int pivot_idx = pivot * N + pivot;
        int current_idx = pivot * N + pivot + tid;
        float scale = MAT[pivot_idx];
        __syncthreads();
        MAT[current_idx] /= scale;
    }
}

__global__
void subtract_single_row(float *MAT, int row, int pivot)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < N)
    {
        int pivot_start_idx = pivot * N + pivot;
        int row_start_idx = row * N + pivot;
        int current_idx = row * N + tid;
        int pivot_idx = pivot * N + tid;

        float f = MAT[row_start_idx] / MAT[pivot_start_idx];
        MAT[current_idx] -= f * MAT[pivot_idx];
    }
}

__global__
void subtract_rows(float *MAT, int pivot)
{
    int tid = blockIdx.y * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    /* Index of element to be modified */
    int row = pivot + 1 + (tid / (N - pivot));
    int col = pivot + (tid % (N - pivot));

    if (row < M && col < N)
    {
        int idx = row * N + col;

        /* Index of leading coefficient of current row */
        int start_idx = row * N + pivot;

        /* Index of pivot leading coefficient */
        int pivot_idx = pivot * N + pivot;

        /* Index of pivot row element value of the same column */
        int source_idx = pivot * N + col;

        /* Get scaling factor */
        float f = MAT[start_idx] / MAT[pivot_idx];

        /* Subtract */
        MAT[idx] -= f * MAT[source_idx];
    }
}

double ref_cuda(float *MAT)
{

    /* Allocate memory for the device */
    float *MATD;
    hipMalloc(&MATD, sizeof(float) * M * N);
    hipMemcpy(MATD, (void *) MAT, sizeof(float) * M * N, hipMemcpyHostToDevice);

    struct timeval begin, end;
    gettimeofday(&begin, 0);

    /* Loop through rows */
    for (int row = 0; row < M; row++)
    {
        /* Assuming square positive matrix always */
        int col = row;

        /* Block size is the number of blocks required 
         * 
         * So if we have 32 elements in the row and BLOCK_SIZE is 16, we need
         * 2 blocks to process this row
         *
         * Optimization can be made since we don't need to process
         * all the elements before the pivot, since
         * we can assume they're zeros
         *
         * Normally we just do M / BLOCK_SIZE
         * But this is modified so that we always have at least 1 block
         */
        int elements_to_process = N - col;
        int num_blocks = (int) ceil((float) elements_to_process / BLOCK_SIZE);
        scale_row<<<num_blocks, BLOCK_SIZE>>>(MATD, row);
        hipDeviceSynchronize();

        #ifdef DEBUG_GPU
        printf("Scaling row %d\n", row);
        hipMemcpy(MAT, (void *) MATD, sizeof(float) * M * N, hipMemcpyDeviceToHost);
        print_mat(MAT);
        #endif

        elements_to_process = (N - col) * (M - row - 1);
        num_blocks = (int) ceil((float) elements_to_process / BLOCK_SIZE);
        subtract_rows<<<num_blocks, BLOCK_SIZE>>>(MATD, row);
        hipDeviceSynchronize();

        #ifdef DEBUG_GPU
        printf("Eliminating rows after row %d\n", row);
        hipMemcpy(MAT, (void *) MATD, sizeof(float) * M * N, hipMemcpyDeviceToHost);
        print_mat(MAT);
        #endif
    }
    gettimeofday(&end, 0);
    
    /* Copy back from device to host */
    hipMemcpy(MAT, (void *) MATD, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    /* Free device memroy */
    hipFree(MATD);

    double time_parallel = (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1e-6;
    return time_parallel;
}

int main(void)
{
    /* Time keeping */
    struct timeval begin, end;
    double time_serial;

    /* Malloc matrices */
    MAT   = (float *) malloc(sizeof(float) * N * M);
    MAT_B = (float *) malloc(sizeof(float) * N * M);

    printf("(M x N)=(%d x %d)\n", M, N);
    init_array(MAT);
    memcpy(MAT_B, MAT, sizeof(float) * N * M);
    print_mat(MAT);

    /* Run single threaded */
    #ifndef CUDA_ONLY
    printf("Running serial . . .\n");
    gettimeofday(&begin, 0);
    ref_old_noswap(MAT_B);
    gettimeofday(&end, 0);
    time_serial = (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1e-6;
    print_mat(MAT_B);
    #endif

    /* Run parallel ref */
    printf("Running parallel . . .\n");
    ref_cuda(MAT);
    double time_parallel = ref_cuda(MAT);

    /* Run verification (if enabled) */
    #ifndef CUDA_ONLY
    #ifdef RUN_VERIF
    printf("Running verification . . .\n");
    int errors = verify_ref(MAT, MAT_B);
    printf("MISMATCH=%d\n", errors);
    #endif
    #endif

    #ifndef CUDA_ONLY
    printf("SERIAL TIME=%.6e s\n", time_serial);
    #endif
    
    printf("PARALL TIME=%.6e s\n", time_parallel);

    return 0;
}
