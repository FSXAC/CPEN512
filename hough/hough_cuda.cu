#include "hip/hip_runtime.h"
#include "common.h"

#define BLOCK_SIZE 1024

__global__
void acc_vote(uint8_t *img, float *acc, int acc_width, int acc_height)
{
    int thread_id = blockIdx.y * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    int row = thread_id / acc_width;
    int col = thread_id % acc_width;

    /* If within bound of the output space */
    if (row < acc_height && col < acc_width)    
    {
        int r = row + MIN_R;
        float theta = MIN_THETA + (D_THETA * col);
        
        if (theta < -(PI / 4) || theta > PI / 4)
        {
            for (int x = 0; x < IMG_SIZE; x++)
            {
                int y = (r - cos(theta) * x) / sin(theta);
                if (IN_BOUND(x, y))
                    GETACC(acc, row, col) += GETIM(img, x, y) > THRESHOLD;
            }
        }
        else
        {
            for (int y = 0; y < IMG_SIZE; y++)
            {
                int x = (r - sin(theta) * y) / cos(theta);
                if (IN_BOUND(x, y))
                    GETACC(acc, row, col) += GETIM(img, x, y) > THRESHOLD;
            }
        }
    }
}

double hough_cuda(uint8_t *img, float *acc, int acc_width, int acc_height)
{
    struct timeval begin, end;
    // gettimeofday(&begin, 0);
    
    /* allocate device memory */
    float *device_img;
    float *device_acc;
    
    hipMalloc(&device_img, sizeof(uint8_t) * IMG_SIZE * IMG_SIZE);
    hipMalloc(&device_acc, sizeof(float) * acc_width * acc_height);
    
    /* Copy input image file */
    hipMemcpy(device_img, (void *) img, sizeof(uint8_t) * IMG_SIZE * IMG_SIZE, hipMemcpyHostToDevice);
    
    gettimeofday(&begin, 0);
    int num_output_elements = acc_width * acc_height;
    acc_vote<<<num_output_elements, BLOCK_SIZE>>>(img, acc, acc_width, acc_height);
    hipDeviceSynchronize();

    /* Get the output */
    hipMemcpy(acc, (void *) device_acc, sizeof(float) * acc_width, acc_height, hipMemcpyDeviceToHost);

    /* Free device memory */
    hipFree(device_img);
    hipFree(device_acc);

    gettimeofday(&end, 0);
    return TIME(begin, end);
}

int main() {

    /* Read image */
    int width, height, bpp;
    uint8_t* bin_image = stbi_load(IMG_FILE, &width, &height, &bpp, 1);
    printf("Image size: %d px by %d px, bpp: %d\n", width, height, bpp);
    
    if (width != IMG_SIZE || height != IMG_SIZE)
    {
        printf("Error! invalid image size\n");
        return 1;
    }

    /* Set up accumulator */
    /* Using float because int could run into overflow issues */
    int acc_width = THETA_STEPS;
    int acc_height = 2 * MAX_R;
    float *acc = (float *) malloc(sizeof(float) * acc_height * acc_width);

    // For each radius
    printf("transforming to %d by %d acc...\n", acc_width, acc_height);
    double t = hough_cuda(bin_image, acc, acc_width, acc_height);
    printf("Execution time: %.2f s\n", t);
    
    /* Normalize and out */
    uint8_t* out_acc = (uint8_t *) malloc(sizeof(uint8_t) * acc_height * acc_width);
    printf("normalizing and copying output...\n");
    float max = normalize_image(acc, out_acc, acc_width, acc_height);
    printf("maximum value in acc: %.1f\n", max);

    /* Write out image to file */
    stbi_write_jpg("out.jpg", acc_width, acc_height, 1, out_acc, 90);

    /* Close image */
    stbi_image_free(bin_image);

    /* Free memory */
    free(out_acc);
    free(acc);

    return 0;
}
